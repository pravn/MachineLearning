#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include "sampler.h"
#include <fstream>
#include <hip/hip_runtime.h>

void sampler::generate_samples(){
  generate_gmm_data();
  std::vector<float2> gmm_data = get_gmm_data();
  size_t c=0;


  for_each(gmm_data.begin(), gmm_data.end(), 
	   [this, &c](float2 t){
	     float epsilon = 100.0f;
	     std::vector<float> &data = this->data;
	     
	     int n = this->num_points*t.y;
	     //	     std::cout << t.y << std::endl;
	     auto x = t.x;
	     std::fill(data.begin()+c, data.begin()+c+n, x);
	     c +=n;
	     histogram.push_back(make_float2(x, n));

	   });

}

void sampler::print_samples(){
  for(int i=0; i<data.size(); i++){
    std::cout << data[i] << std::endl;
  }
}

void sampler::write_samples(){
  std::ofstream fout;
  fout.open("samples.txt");
  for(int i=0; i<data.size(); i++){
    fout << data[i] << std::endl;
  }
  fout.close();

  fout.open("histogram.txt");
  std::vector<float2>::iterator it;
  for(it=histogram.begin(); it!=histogram.end(); it++){
    fout << it->x << " " << it->y << std::endl;
  }
  fout.close();

}
    

