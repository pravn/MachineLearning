#include "hip/hip_runtime.h"
#include "utils.h"
inline float pi() { return std::atan(1)*4.0;}

  float gaussian::operator()(float y){
    return 1.0/sqrtf(2.0f*pi()*sigma*sigma)*exp(-(y-mu)*(y-mu)/(2*sigma*sigma));
   }

float random_number(){
  float u = rand();
  return u/(float) RAND_MAX;
};
