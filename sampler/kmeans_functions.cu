#include "hip/hip_runtime.h"
#include "gmm.h"
#include "sampler.h"
#include <vector> 
#include <cassert>
#include <iostream>
#include <algorithm>

void zero_out(std::vector<float> mu, std::vector<float> mu_new, std::vector<int> count, size_t num_components){
  assert((mu.size()==count.size())&&(mu.size()==num_components));
  for(size_t i=0; i<num_components; i++){
    mu[i] = 0.0f;
    mu_new[i] = 0.0f;
    count[i] = 0;
  }
}
    

void init_mu(std::vector<float> &mu, std::vector<float> data){
  float y = 0;
  for_each(data.begin(), data.end(), [&y](float t){
      y = (y>t)?y:t;
    });

  std::cout << "max " << y << std::endl;
      
  
 for_each(mu.begin(), mu.end(), [y](float &mu){
      mu = y * rand()/(float) RAND_MAX;
      });

  /*     mu[0] = 0.5;
	 mu[1] = 0.6;*/
    
  std::cout << "mu[0], mu[1] " << mu[0] <<  " " << mu[1] << std::endl;
  
}


inline float squared_distance(float x, float mu){
  return (x-mu)*(x-mu);
}

//iterative estimates of cluster means
//naturally, we will have to redo all this when we get to CUDA
void iterate_mu(float x, std::vector<float> mu, std::vector<float> &mu_new, std::vector<int> &count){
  float J=100.0;
  int i_k=0;
  float D;

  for(int i=0; i<mu.size(); i++){
    if((D = squared_distance(x,mu[i]))<J){
    
      J = D;
      i_k = i;

    }

  }


  count[i_k]++;
  mu_new[i_k] += x;

 }


void sampler::cluster_kmeans(){
  //initialize r_nk, mu_k
  std::cout << "in cluster_kmeans" << std::endl;
  float J=0; //distance/cost function
  std::vector<float>mu(num_components);
  std::vector<int> count(num_components);
  std::vector<float>mu_new(num_components);

  zero_out(mu, mu_new, count, num_components);
  init_mu(mu, data);

  for(int i=0; i<num_components; i++){
    std::cout << mu[i] << " r  " << mu_new[i] << std::endl; 
  }
  
  std::vector<float>::iterator it;

  for(int j=0; j<10; j++){

    std::fill(mu_new.begin(), mu_new.end(), 0);
    std::fill(count.begin(), count.end(), 0);

  for(it=data.begin(); it!=data.end(); it++){
    iterate_mu(*it, mu, mu_new, count);
  }

  for(int i=0; i<num_components; i++){
    mu_new[i] = mu_new[i]/count[i];
  }

  mu = mu_new;
  std::cout << mu_new[0] << " s " << mu_new[1] << " c " << count[0] << " c " << count[1] << std::endl;  

  }

  std::cout << "Done with cluster_kmeans" << std::endl;
}
  
 
