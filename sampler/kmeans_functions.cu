#include "hip/hip_runtime.h"
#include "gmm.h"
#include "sampler.h"
#include <vector> 
#include <cassert>
#include <iostream>
#include <algorithm>
#include "utils.h"

void zero_out(std::vector<float> mu, std::vector<float> mu_new, std::vector<int> count, size_t num_components){
  assert((mu.size()==count.size())&&(mu.size()==num_components));
  for(size_t i=0; i<num_components; i++){
    mu[i] = 0.0f;
    mu_new[i] = 0.0f;
    count[i] = 0;
  }
}
    

void init_mu(std::vector<float> &mu, std::vector<float> data){
  float y = 0;
  for_each(data.begin(), data.end(), [&y](float t){
      y = (y>t)?y:t;
    });

  std::cout << "max " << y << std::endl;
      
  
 for_each(mu.begin(), mu.end(), [y](float &mu){
      mu = y * rand()/(float) RAND_MAX;
      });

  /*     mu[0] = 0.5;
	 mu[1] = 0.6;*/
    
  std::cout << "mu[0], mu[1] " << mu[0] <<  " " << mu[1] << std::endl;
  
}


inline float squared_distance(float x, float mu){
  return (x-mu)*(x-mu);
}

//iterative estimates of cluster means
//naturally, we will have to redo all this when we get to CUDA
void iterate_mu(float x, std::vector<float> mu, std::vector<float> &mu_new, std::vector<float> &sigma, std::vector<int> &count){
  float J=100.0;
  int i_k=0;
  float D;

  for(int i=0; i<mu.size(); i++){
    if((D = squared_distance(x,mu[i]))<J){
    
      J = D;
      i_k = i;

    }

  }


  count[i_k]++;
  mu_new[i_k] += x;
  sigma[i_k] += J;

 }


void sampler::cluster_kmeans(){
  //initialize r_nk, mu_k
  std::cout << "in cluster_kmeans" << std::endl;
  float J=0; //distance/cost function
  //  std::vector<float>mu(num_components);
  //  std::vector<int> count(num_components);
  std::vector<float>mu_new(num_components);

  zero_out(mu, mu_new, count, num_components);
  init_mu(mu, data);

  for(int i=0; i<num_components; i++){
    std::cout << mu[i] << " r  " << mu_new[i] << std::endl; 
  }
  
  std::vector<float>::iterator it;

  for(int j=0; j<10; j++){

    std::fill(mu_new.begin(), mu_new.end(), 0);
    std::fill(count.begin(), count.end(), 0);
    std::fill(sigma.begin(), sigma.end(), 0);

  for(it=data.begin(); it!=data.end(); it++){
    iterate_mu(*it, mu, mu_new, sigma, count);
  }

  for(int i=0; i<num_components; i++){
    mu_new[i] = mu_new[i]/count[i];
  }

  mu = mu_new;

  for(int i=0; i<num_components; i++){
    sigma[i] = sigma[i]/count[i];
  }


  std::cout << " m " <<  mu_new[0] << " m " << mu_new[1] << " s " << sqrtf(sigma[0]) << " s " << sqrtf(sigma[1]) << " c " << count[0] << " c " << count[1] << std::endl;  


  }

  std::cout << "Done with cluster_kmeans" << std::endl;
}
  

void sampler::train_gmm(){
  //Assume two clusters 
  //step 1 
  //compute responsibilities 
  //gamma(z_nk) = w_k N(x_n|mu_k, sigma_k)/(sum_j w_j N(x_n|mu_j, sigma_j)


  std::cout << "Starting training gmm " << std::endl;

  float *gamma_nk;
  float *mu_k;
  float *sigma_k;
  float *weights_k;
  float *N_k;
  float *sum_gamma;


  gamma_nk = new float[num_components*num_points];
  mu_k = new float[num_components];
  sigma_k = new float[num_components];
  weights_k = new float[num_components];
  N_k = new float[num_components];
  sum_gamma = new float[num_points];


  //need to assume mu, sigma;
  /*  for(int i=0; i<num_components; i++){
    mu_k[i] = rand()/ (float) RAND_MAX;
    sigma_k[i] = rand()/(float) RAND_MAX;
    weights_k[i] = rand()/ (float) RAND_MAX;
    }*/


  mu_k[0] = 0.1;
  sigma_k[0] = 0.016;
  mu_k[1] = 0.6;
  sigma_k[1] = 0.55;

  weights_k[0] = 0.5;
  weights_k[1] = 0.5;
				      

  //normalize weights;
  float marginal = 0;
  for (int i=0; i<num_components; i++){
    marginal += weights_k[i];
  }


  for(int i=0; i<num_components; i++){
    weights_k[i]/=marginal;
  }

  //begin iterations
  for(int iter = 0; iter<200; iter++){


    for(int i=0; i<num_points; i++){
      sum_gamma[i] = 0;
    }

  //compute responsibilities;
  for(int i=0; i<num_points; i++){


    for(int j=0; j<num_components; j++){
      gaussian g(mu_k[j], sqrtf(sigma_k[j]));
      float y = data[i];
      gamma_nk[i*num_components+j] = weights_k[j]*g(y);
      sum_gamma[i] += weights_k[j]*g(y);


    }
  }

  for(int i=0; i<num_points; i++){
    for(int  j=0; j<num_components; j++){
            gamma_nk[i*num_components+j]/=sum_gamma[i];
    }
  }


  for(int i=0; i<num_components; i++){
    N_k[i] = 0;
  }


  for(int i=0; i<num_points; i++){
    for(int j=0; j<num_components; j++){
      N_k[j] += gamma_nk[i*num_components+j];
    }
  }

  for(int i=0; i<num_components; i++){
    mu_k[i] = 0;
  }


  for(int i=0; i<num_points; i++){
    for(int j=0; j<num_components; j++){
      mu_k[j] += 1.0/N_k[j] * gamma_nk[i*num_components+j]*data[i];
    }
  }

  for(int i=0; i<num_components; i++){
    sigma_k[i] = 0;
  }



  for(int i=0; i<num_points; i++){
    for(int j=0; j<num_components; j++){
      sigma_k[j] += 1.0/N_k[j]*gamma_nk[i*num_components+j]*(data[i]-mu_k[j])*(data[i]-mu_k[j]);
    }
  }

  for(int j=0; j<num_components; j++){
    weights_k[j] = N_k[j]/num_points;
  }

  std::cout << " N " <<  N_k[0] << " N " << N_k[1] << " N " << num_points << std::endl;
  std::cout << " w " <<  weights_k[0] << " w " << weights_k[1] << std::endl;
  std::cout << " m " << mu_k[0] << " m " << mu_k[1] << std::endl;
  std::cout << " s " << sqrtf(sigma_k[0]) << " s " << sqrtf(sigma_k[1]) << std::endl;

  }

  

  

  std::cout << "Done training gmm " << std::endl;
  
  delete[] gamma_nk;
  delete[] mu_k;
  delete[] sigma_k;
  delete[] N_k;
  delete[] weights_k;
  delete[] sum_gamma;
}
    

    


  

    


  
  
