#include <iostream>
#include <vector>
#include <algorithm>
#include "sampler.h"
#include <fstream>
#include <hip/hip_runtime.h>


int main(){
  // ask for 100000 points to be sampled from distribution
  // we create a mixture distribution consisting of two gaussian components
  sampler S(100000, 1000, 2);
  S.generate_samples();
  S.write_samples();
  S.write_gmm_data();
  S.cluster_kmeans();

}



