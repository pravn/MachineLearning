#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <algorithm>
#include <iostream>
#include <cstdlib>

#include <fstream>
#include <iomanip>
#include "gmm.h"

//make const
inline float pi() { return std::atan(1)*4.0;}

struct gaussian{
  gaussian(float _mu, float _sigma): mu(_mu), sigma(_sigma){};

  float operator()(float y){
    return 1.0/sqrtf(2.0f*pi()*sigma*sigma)*exp(-(y-mu)*(y-mu)/(2*sigma*sigma));
   }

  float mu;
  float sigma;
};


float random_number(){
  float u = rand();
  return u/(float) RAND_MAX;
};


void GMM::generate_gmm_data(){
  gaussian u = gaussian(0.7, 0.4);
  std::vector<float2>::iterator it;
  size_t count = 0;

  for(it=data.begin(),count=0; count<size/2; it++, count++){
    it->x = rand()/(float) RAND_MAX;
    it->y = u(it->x);
  }

  gaussian v = gaussian(0.2, 0.1);
  for(;it!=data.end(); it++){
    it->x = rand()/(float) RAND_MAX;
    it->y = v(it->x);
  }

  //normalize
  float marginal = 0;
  for_each(data.begin(), data.end(), 
	   [&marginal](float2 t){
	     marginal +=t.y;
	     });


  for_each(data.begin(), data.end(),[marginal](float2 &t){
      t.y/=marginal;
    });

}

void GMM::print_gmm_data(){
  std::vector<float2>::iterator it;
  for(it=data.begin(); it!=data.end(); it++){
    std::cout << it->x << " " << it->y << std::endl;
  }

}

void GMM::write_gmm_data(){
  std::vector<float2>::iterator it;
  std::ofstream fout;
  fout.open("data.txt");
  for(it=data.begin(); it!=data.end(); it++){
    fout << it->x << " " << it->y << std::endl;
  }
  fout.close();
}
  


/*
int main(){
  const int num_components=2;
  const int num_points = 1000;
  GMM gmm(num_points, num_components);
    gmm.generate_gmm_data();
    //    gmm.print_data();
    gmm.write_gmm_data();
    //    std::cout << "yes yes " << std::endl;
    //    gmm.cluster_kmeans();
  }
  */
 

