#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <algorithm>
#include <iostream>
#include <cstdlib>

#include <fstream>
#include <iomanip>
#include "gmm.h"
#include "utils.h"

void GMM::generate_gmm_data(){
  gaussian u = gaussian(0.2, 0.1);
  std::vector<float2>::iterator it;
  size_t count = 0;

  for(it=data.begin(),count=0; count<size/2; it++, count++){
    it->x = 0 + (0.5)*rand()/(float) RAND_MAX;
    it->y = u(it->x);
  }

  gaussian v = gaussian(0.8, 0.05);
  for(;it!=data.end(); it++){
    it->x = 0.5+ (1-0.5)*rand()/(float) RAND_MAX;
    it->y = v(it->x);
  }

  //normalize
  float marginal = 0;
  for_each(data.begin(), data.end(), 
	   [&marginal](float2 t){
	     marginal +=t.y;
	     });


  for_each(data.begin(), data.end(),[marginal](float2 &t){
      t.y/=marginal;
    });

}

void GMM::print_gmm_data(){
  std::vector<float2>::iterator it;
  for(it=data.begin(); it!=data.end(); it++){
    std::cout << it->x << " " << it->y << std::endl;
  }

}

void GMM::write_gmm_data(){
  std::vector<float2>::iterator it;
  std::ofstream fout;
  fout.open("data.txt");
  for(it=data.begin(); it!=data.end(); it++){
    fout << it->x << " " << it->y << std::endl;
  }
  fout.close();
}
  


/*
int main(){
  const int num_components=2;
  const int num_points = 1000;
  GMM gmm(num_points, num_components);
    gmm.generate_gmm_data();
    //    gmm.print_data();
    gmm.write_gmm_data();
    //    std::cout << "yes yes " << std::endl;
    //    gmm.cluster_kmeans();
  }
  */
 

